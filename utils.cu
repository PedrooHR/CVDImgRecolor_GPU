#include "hip/hip_runtime.h"

#include "utils.h"
#include <stdio.h>


float Proto[3][3] = {{0.152286, 1.052583, -0.204868},
                    {0.114503, 0.786281, 0.099216},
                    {-0.003882, -0.048116, 1.051998}};

float Deuto[3][3] = {{0.367322, 0.860646, -0.227968},
                    {0.280085, 0.672501, 0.047413},
                    {-0.011820, 0.042940, 0.968881}};

float Trito[3][3] = {{1.255528, -0.076749, -0.178779},
                    {-0.078411, 0.930809, 0.147602},
                    {0.004733, 0.691367	, 0.303900}};


#define maximo(a,b) a > b ? a : b
#define minimo(a,b) a < b ? a : b

float NDcoeff = (1.0 / sqrtf(2*M_PI*SIGMA*SIGMA));
float NDmax   = NDcoeff * pow(euler, -((MI*MI) / (2*SIGMA*SIGMA)));

std::vector <float> getRGBColor(float L, float a, float b){
	float X_ref =  94.811 / 100.0; // D65 Observer - Daylight, sRGB, Adobe-RGB
	float Y_ref = 100.000 / 100.0;
	float Z_ref = 107.304 / 100.0;

	float fy = (L + 16.0) / 116.0,
		  fx = (a / 500.0) + fy,
		  fz = fy - (b / 200.0),
		  xr, yr, zr,
		  X, Y, Z,
		  r, g, B;

	xr = (fx * fx * fx > 0.008856) ? fx * fx * fx : ((116.0 * fx) - 16.0) / 903.3;
	yr = (L > (903.3 * 0.008856)) ? fy * fy * fy : L / 903.3;
	zr = (fz * fz * fz > 0.008856) ? fz * fz * fz : ((116.0 * fz) - 16.0) / 903.3;

	X = xr * X_ref;
	Y = yr * Y_ref;
	Z = zr * Z_ref;

	r = X *  3.2406 + Y * -1.5372 + Z * -0.4986;
	g = X * -0.9689 + Y *  1.8758 + Z *  0.0415;
	B = X *  0.0557 + Y * -0.2040 + Z *  1.0570;

	std::vector <float> RGB = {r, g, B};

	return RGB;
}

std::vector <float> getLabColor(unsigned int sR, unsigned int sG, unsigned int sB)
{
	float X_ref =  94.811; // D65 Observer - Daylight, sRGB, Adobe-RGB
	float Y_ref = 100.000;
	float Z_ref = 107.304;

	float r = sR / 255.0,
		  g = sG / 255.0,
		  b = sB / 255.0,
		  x,y,z,
	      fx, fy, fz;

	//RGB to XYZ
	x = (r * 0.4124 + g * 0.3576 + b * 0.1805) / (X_ref/100.0);
	y = (r * 0.2126 + g * 0.7152 + b * 0.0722) / (Y_ref/100.0);
	z = (r * 0.0193 + g * 0.1192 + b * 0.9505) / (Z_ref/100.0);

	//XYZ to Lab
	fx = (x > 0.008856) ? pow(x, 1/3.0) : (7.787 * x) + 16.0/116.0;
	fy = (y > 0.008856) ? pow(y, 1/3.0) : (7.787 * y) + 16.0/116.0;
	fz = (z > 0.008856) ? pow(z, 1/3.0) : (7.787 * z) + 16.0/116.0;

	std::vector <float> v = {(116 * fy) - 16,
							 500 * (fx - fy),
							 200 * (fy - fz)};

	return v;
}

float sqED_2D(float x1, float y1, float x2, float y2){
	float dist = (x1 - x2) * (x1 - x2);
	dist += (y1 - y2) * (y1 - y2);
	return dist;
}


