#include "hip/hip_runtime.h"
/*
 * grid.cpp
 *
 *  Created on: 17/03/2018
 *      Author: phr
 */

#include "grid.h"
#include <iostream>
#include "utils.h"

Grid::Grid(int type, int graph_size) {
	GRAPH_SIZE = graph_size;
	numEdges = GRAPH_SIZE - 1;
	numRibs = GRAPH_SIZE - 2;
	hipHostMalloc((void**) &Graph, GRAPH_SIZE * sizeof(Node));
	// do graphnodes
	if (type == PROTANOPE) {
		//PROTANOPE LIMITS
		A = {8.648425, -73.086372, 56.664734};
		C = {-14.907598, 86.293831, 89.536812};

		miAB = A[1] / A[0];
		miBC = C[1] / C[0];

		float x_step = fabs((A[0] - C[0]) / (GRAPH_SIZE * 1.0));
		float x_start = C[0];

		for(int i = 0; i < GRAPH_SIZE; i++) {
			Graph[i].id = i;
			Graph[i].Position = (float*) malloc (2*sizeof(float));
			Graph[i].Position[0] = (x_start <= 0) ? miBC * x_start : miAB * x_start;
			Graph[i].Position[1] = -x_start;
			Graph[i].CVDposition = (float*) malloc (2*sizeof(float));
			Graph[i].CVDposition[0] = x_start;
			Graph[i].CVDposition[1] = (x_start <= 0) ? miBC * x_start : miAB * x_start;
			Graph[i].Weight = 1.0f;
			Graph[i].a = (x_start <= 0) ? miBC * x_start : miAB * x_start;
			Graph[i].b = -x_start;
			x_start += x_step;
		}
	}

	// do edges
	for (int i = 0; i < GRAPH_SIZE - 1; i++) {
		std::vector<int> t = { i, i + 1 };
		Edges.push_back(t);
	}

	// do ribs
	for (int i = 0; i < GRAPH_SIZE - 2; i++) {
		std::vector<int> t = { i + 1, i, i + 2 };
		Ribs.push_back(t);
	}
}

