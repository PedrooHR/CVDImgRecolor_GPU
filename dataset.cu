/*
 * dataset.cpp
 *
 *  Created on: 17/03/2018
 *      Author: phr
 */

#include "dataset.h"
#include "utils.h"

Dataset::Dataset(const char *imgPath){
	int local_id = 0;
	cimg_library::CImg<unsigned int> image(imgPath);
	width = image.width();
	height = image.height();
	int size = width * height;

	hipHostMalloc((void**)&Datapoints, size * sizeof(Pixel));
	cimg_forXY(image,x,y) {
		Datapoints[local_id].id = local_id;
		Datapoints[local_id].closest_node = -1;
		//create RGB
		Datapoints[local_id].RGB = (float*) malloc(3* sizeof(float));
		Datapoints[local_id].R = Datapoints[local_id].RGB[0] = image(x,y,0)/255.0f;
		Datapoints[local_id].G = Datapoints[local_id].RGB[1] = image(x,y,1)/255.0f;
		Datapoints[local_id].B = Datapoints[local_id].RGB[2] = image(x,y,2)/255.0f;
		//create Lab
		std::vector <float> lab_color = getLabColor(image(x,y,0), image(x,y,1), image(x,y,2));
		Datapoints[local_id].Lab = (float*) malloc(3* sizeof(float));
		Datapoints[local_id].L = Datapoints[local_id].Lab[0] = lab_color[0];
		Datapoints[local_id].a = Datapoints[local_id].Lab[1] = lab_color[1];
		Datapoints[local_id].b = Datapoints[local_id].Lab[2] = lab_color[2];

		local_id++;
	};
	Datasize = size;
}
