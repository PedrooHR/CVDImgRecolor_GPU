#include "hip/hip_runtime.h"
/*
 * solver.cpp
 *
 *  Created on: 17/03/2018
 *      Author: phr
 */

#include <thrust/device_ptr.h>
#include "solver.h"
#include "grid.h"
#include "dataset.h"

#define Min2(x,y) x < y ? x : y
#define Min3(x,y,z) Min2(x,Min2(y,z))
#define Max2(x,y) x > y ? x : y
#define Max3(x,y,z) Max2(x,Max2(y,z))

Solver::Solver(Grid *A, Dataset *B){
	inverte = false;
	grid = A;
	dataset = B;
	numEpochs = 7;
	zeroNode = -1;
	Emodule = 0.1;
	Rmodule = 50;
	CurrentEpoch = 0;
	GridDimension = 2;
	numNodes = A->GRAPH_SIZE;
	SysMatrix.resize(numNodes);
	Taxons.resize(numNodes);
	ResultVector.resize(numNodes);
	Solution.resize(numNodes);
	NodesSolution.resize(dataset->Datasize);
	for (int i = 0; i < numNodes; i++){
		SysMatrix[i].resize(numNodes);
		Solution[i].resize(2);
		ResultVector[i].resize(2);
	}
	float epo = 0.05;
	for (int i = 0; i < numEpochs; i++){
		EpochM.push_back(epo);
		epo /= (3 - i*0.04);
	}
}

void Solver::resetSysMatrix(){
	// Clear stage
	SysMatrix.clear();
	Taxons.clear();
	ResultVector.clear();
	Solution.clear();

	//Resize stage
	SysMatrix.resize(numNodes);
	Taxons.resize(numNodes);
	ResultVector.resize(numNodes);
	Solution.resize(numNodes);
	for (int i = 0; i < numNodes; i++){
		SysMatrix[i].resize(numNodes);
		Solution[i].resize(2);
		ResultVector[i].resize(2);
	}
}

void Solver::constructSysMatrix(){
	resetSysMatrix();

	Emodule = EpochM[CurrentEpoch]*pow(grid->numEdges, (2 - GridDimension)/GridDimension)*10;
	Rmodule = EpochM[CurrentEpoch]*pow(grid->numRibs, (2 - GridDimension)/GridDimension)*100;

	calcTaxons();

 	// Funcional
	for (int i = 0; i < grid->numRibs; i++){
		int N0 = grid->Ribs[i][0];
		int N1 = grid->Ribs[i][1];
		int N2 = grid->Ribs[i][2];

		SysMatrix[N0][N0] += Rmodule * 4.0;
		SysMatrix[N0][N1] -= Rmodule * 2.0;
		SysMatrix[N0][N2] -= Rmodule * 2.0;
		SysMatrix[N1][N0] -= Rmodule * 2.0;
		SysMatrix[N2][N0] -= Rmodule * 2.0;
		SysMatrix[N1][N1] += Rmodule;
		SysMatrix[N1][N2] += Rmodule;
		SysMatrix[N2][N1] += Rmodule;
		SysMatrix[N2][N2] += Rmodule;
	}

	for (int i = 0; i < grid->numEdges; i++){
		int N0 = grid->Edges[i][0];
		int N1 = grid->Edges[i][1];

		SysMatrix[N0][N0] += Emodule;
		SysMatrix[N1][N1] += Emodule;
		SysMatrix[N0][N1] -= Emodule;
		SysMatrix[N1][N0] -= Emodule;
	}

	for (int i = 0; i < numNodes; i++){
		SysMatrix[i][i] += Taxons[i].size() / (1.0*dataset->Datasize);
	}

	for (int i = 0; i < numNodes; i++){
		ResultVector[i][0] = 0;
		ResultVector[i][1] = 0;
		for (int j = 0; j < Taxons[i].size(); j++){
			ResultVector[i][0] += dataset->Datapoints[Taxons[i][j]].Lab[1];
			ResultVector[i][1] += dataset->Datapoints[Taxons[i][j]].Lab[2];
		}
		ResultVector[i][0] /= (1.0*dataset->Datasize);
		ResultVector[i][1] /= (1.0*dataset->Datasize);
	}
}

void Solver::solveLS(){
	//escalonador
	for (int k = 0; k < numNodes; k++) {
		for (int j = k + 1; j < numNodes; j++){
			float multi = SysMatrix[j][k] / SysMatrix[k][k];
			for (int i = k; i < numNodes; i++)
				SysMatrix[j][i] -= (SysMatrix[k][i] * multi);

			ResultVector[j][0] -= ResultVector[k][0] * multi;
			ResultVector[j][1] -= ResultVector[k][1] * multi;
		}
	}
	//resolve
	for (int k = numNodes-1; k >= 0; k--) {
		double somax = 0;
		double somay = 0;
		for (int j = k + 1; j < numNodes; j++){
			somax += SysMatrix[k][j] * Solution[j][0];
			somay += SysMatrix[k][j] * Solution[j][1];
		}
		Solution[k][0] = (ResultVector[k][0] - somax) / SysMatrix[k][k];
		Solution[k][1] = (ResultVector[k][1] - somay) / SysMatrix[k][k];
	}

	for (int i = 0; i < numNodes; i++){
			grid->Graph[i].Position[0] = Solution[i][0];
			grid->Graph[i].Position[1] = Solution[i][1];
	}
}

__global__ void calcTaxonI (Pixel *Datapoints, Node *Graph, int size){
	int i = threadIdx.x + (blockDim.x * blockIdx.x);
	if (i < size) {
		float MinDist = (float) MAX_VALUE;
		int MinNodeRef;
		for (int j = 0; j < T_GRID; j++) {
			float localDist = (Datapoints[i].a - Graph[j].a)
					* (Datapoints[i].a - Graph[j].a);
			localDist += (Datapoints[i].b - Graph[j].b)
					* (Datapoints[i].b - Graph[j].b);

			if (localDist < MinDist) {
				MinDist = localDist;
				MinNodeRef = j;
			}
		}
		Datapoints[i].closest_node = MinNodeRef;
	}
}

void Solver::calcTaxons(){
	for (int i = 0; i < numNodes; i++)
		Taxons[i].clear();

	int div = (dataset->Datasize / 1024) + 1;
	calcTaxonI <<<div, 1024>>> (dataset->Datapoints, grid->Graph, dataset->Datasize);

	hipDeviceSynchronize();

	for (int i = 0; i < dataset->Datasize; i++){
		Taxons[dataset->Datapoints[i].closest_node].push_back(i);
	}
}

void Solver::invert(){
	float sumDistAB = 0, sumDistBA = 0;
		for(int i = 0; i < Taxons.size(); i++){ //para cada nodo
			for (int j = 0; j < Taxons[i].size(); j++){

				float Deuto[3][3] = {{0.367322, 0.860646, -0.227968},
				                    {0.280085, 0.672501, 0.047413},
				                    {-0.011820, 0.042940, 0.968881}};
				int current_point = Taxons[i][j];

				std::vector<float> algproj = {grid->Graph[i].CVDposition[0], grid->Graph[i].CVDposition[1], dataset->Datapoints[current_point].Lab[0]};
				std::vector<float> alternativa = {grid->Graph[numNodes - i -1 ].CVDposition[0], grid->Graph[numNodes - i -1 ].CVDposition[1], dataset->Datapoints[current_point].Lab[0]};
				std::vector<float> original = {dataset->Datapoints[current_point].Lab[1], dataset->Datapoints[current_point].Lab[2], dataset->Datapoints[current_point].Lab[0]};

				std::vector <float> RgbProjection;
				RgbProjection.resize(3);
				RgbProjection[0] = std::min(std::max(Deuto[0][0]*original[0] + Deuto[0][1] * original[1] + Deuto[0][2]*original[2],0.0f),1.0f) * 255.0;
				RgbProjection[1] = std::min(std::max(Deuto[1][0]*original[0] + Deuto[1][1] * original[1] + Deuto[1][2]*original[2],0.0f),1.0f) * 255.0;
				RgbProjection[2] = std::min(std::max(Deuto[2][0]*original[0] + Deuto[2][1] * original[1] + Deuto[2][2]*original[2],0.0f),1.0f) * 255.0;

				sumDistAB += sqED_2D(algproj[0], algproj[1], RgbProjection[0], RgbProjection[1]);
				sumDistBA += sqED_2D(alternativa[0], alternativa[1], RgbProjection[0], RgbProjection[1]);
			}
		}
		inverte = false;

		if (sumDistAB > sumDistBA)
			inverte = true;
}

void Solver::centerWhite(){
	float min_distance = (float) MAX_VALUE;
	int zeroNode = numNodes / 2;
	for (int i = 0; i < numNodes; i++){
		for (int j = 0; j < Taxons[i].size(); j++){
			int point = Taxons[i][j];
			float localZeroDist = sqED_2D(dataset->Datapoints[point].a, dataset->Datapoints[point].b, 0, 0);
			if (localZeroDist < min_distance){
				zeroNode = i;
				min_distance = localZeroDist;
			}
		}
	}

	int size_positive = numNodes - zeroNode;
	int size_negative = zeroNode;
	float x_step = fabs((size_positive > size_negative) ? (grid->A[0] / (1.0 * size_positive)) : (-grid->C[0] / (1.0 * size_negative)));

	OriginalMap.resize(numNodes);

	float x_start = 0;
	for (int i = zeroNode; i < numNodes; i++){
		std::vector <float> t = {x_start, grid->miAB * x_start };
		OriginalMap[i] = t;
		x_start += x_step;
	}

	x_start = 0 - x_step;
	for (int i = zeroNode - 1; i >= 0; i--){
		std::vector <float> t = {x_start, grid->miBC * x_start};
		OriginalMap[i] = t;
		x_start -= x_step;
	}
}

void Solver::projectPoints(){
	NodesSolution.clear();
	NodesSolution.resize(dataset->Datasize);
	for(int i = 0; i < Taxons.size(); i++){ //para cada nodo
		for (int j = 0; j < Taxons[i].size(); j++){
			int current_point = Taxons[i][j];
			std::vector<float> algproj, original;

			algproj = {grid->Graph[i].CVDposition[0], grid->Graph[i].CVDposition[1], dataset->Datapoints[current_point].Lab[0]};

			if(OriginalMap.size() > 0){
				algproj = {OriginalMap[i][0], OriginalMap[i][1], dataset->Datapoints[current_point].Lab[0]};
			}

			NodesSolution[current_point].push_back(algproj[0]);
			NodesSolution[current_point].push_back((inverte) ? -algproj[1] : algproj[1]);
			NodesSolution[current_point].push_back(algproj[2]);
		}
	}
}

void Solver::drawRecolored(const char * filepath){
	cimg_library::CImg<float> image(dataset->width, dataset->height, 1, 3, 0);
	for(int i = 0; i < NodesSolution.size(); i++){
		int x = i%dataset->width;
		int y = i/dataset->width;
		std::vector <float> RGBcolor = getRGBColor(NodesSolution[i][2], NodesSolution[i][0], NodesSolution[i][1]);
		for (int j = 0; j < 3; j++){
			RGBcolor[j] = (RGBcolor[j] > 1.0) ? 1.0 : RGBcolor[j];
			RGBcolor[j] = (RGBcolor[j] < 0.0) ? 0.0 : RGBcolor[j];
		}
		image(x,y,0) = RGBcolor[0]*255.0;
		image(x,y,1) = RGBcolor[1]*255.0;
		image(x,y,2) = RGBcolor[2]*255.0;
	}
	image.save_jpeg(filepath, 100);
}
